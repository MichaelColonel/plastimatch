#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------------
   See COPYRIGHT.TXT and LICENSE.TXT for copyright and license information
   ----------------------------------------------------------------------- */
#include "plmutil_config.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_util.h"
#include "distance_map_cuda.h"

__global__ void 
myFirstKernel(int *d_a)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    d_a[idx] = idx;  
}


void
distance_map_cuda (void *dummy_var)
{
    // pointer for host memory
    int *h_a;

    // pointer for device memory
    int *d_a;

    // define grid and block size
    int numBlocks = 8;
    int numThreadsPerBlock = 8;

    // Part 1 of 5: allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc((void **) &d_a, memSize);

    CUDA_check_error("hipMalloc");

    // Part 2 of 5: launch kernel
    dim3 dimGrid(numBlocks, 1, 1);
    dim3 dimBlock(numThreadsPerBlock, 1, 1);

    // Part 3 of 5: implement the kernel
    myFirstKernel<<<dimGrid, dimBlock>>>(d_a);

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    CUDA_check_error("kernel execution");

    // Part 4 of 5: device to host copy
    hipMemcpy( h_a, d_a, memSize, hipMemcpyDeviceToHost );

    // Check for any CUDA errors
    CUDA_check_error("hipMemcpy");

    // Part 5 of 5: verify the data returned to the host is correct
    for (int i = 0; i < numBlocks; i++)	{
	for (int j = 0; j < numThreadsPerBlock; j++) {
	    assert (h_a[i * numThreadsPerBlock + j] == i * numThreadsPerBlock + j);
	}
    }

    // free device memory
    hipFree(d_a);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");
    exit (0);
}
